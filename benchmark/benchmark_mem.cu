//
// Created by depaulsmiller on 9/3/20.
//

#include <StandardSlabDefinitions.h>
#include <vector>
#include <Slab.h>
#include <hip/hip_runtime_api.h>
#include <unordered_map>

using namespace lslab;

namespace lslab {

template<>
struct EMPTY<int *> {
    static constexpr int *value = nullptr;
};

template<>
LSLAB_DEVICE unsigned compare(int *const &lhs, int *const &rhs) {
    return lhs - rhs;
}

}

int main() {

    const int size = 1000;
    std::hash<unsigned> hfn;
    SlabUnified<unsigned, int *> s(size);
    auto b = new BatchBuffer<unsigned, int *>();

    s.setGPU();

    for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
        unsigned j = 0;
        for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
            unsigned key = 1;
            int *value = new int[256]; // allocating 1KB
            for (int w = 0; w < 256; w++) {
                value[w] = 1;
            }
            b->getBatchKeys()[j] = key;
            b->getHashValues()[j] = hfn(key);
            b->getBatchRequests()[j] = REQUEST_INSERT;
            b->getBatchValues()[j] = value;
        }
        for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
            b->getBatchRequests()[j] = REQUEST_EMPTY;
        }
        s.moveBufferToGPU(b, 0x0);
        s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0);
        s.moveBufferToCPU(b, 0x0);
        gpuErrchk(hipStreamSynchronize(0x0));
        j = 0;
        for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
            if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != EMPTY<int *>::value) {
                delete[] b->getBatchValues()[j];
            }
        }
    }

    gpuErrchk(hipProfilerStart());

    for (int rep = 0; rep < 10; rep++) {

        for (unsigned i = 0; i < (unsigned) size; i += THREADS_PER_BLOCK * BLOCKS) {
            unsigned j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS && i * THREADS_PER_BLOCK * BLOCKS + j < size; j++) {
                unsigned key = 1;
                int *value = new int[256]; // allocating 1KB
                for (int w = 0; w < 256; w++) {
                    value[w] = 1;
                }
                b->getBatchKeys()[j] = key;
                b->getHashValues()[j] = hfn(key);
                b->getBatchRequests()[j] = REQUEST_INSERT;
                b->getBatchValues()[j] = value;
            }
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                b->getBatchRequests()[j] = REQUEST_EMPTY;
            }
            s.moveBufferToGPU(b, 0x0);
            s.diy_batch(b, BLOCKS, THREADS_PER_BLOCK, 0x0);
            s.moveBufferToCPU(b, 0x0);
            gpuErrchk(hipStreamSynchronize(0x0));

            j = 0;
            for (; j < THREADS_PER_BLOCK * BLOCKS; j++) {
                if (b->getBatchRequests()[j] == REQUEST_INSERT && b->getBatchValues()[j] != EMPTY<int *>::value) {
                    delete[] b->getBatchValues()[j];
                }
            }
        }
    }

    gpuErrchk(hipProfilerStop());
    delete b;
}
